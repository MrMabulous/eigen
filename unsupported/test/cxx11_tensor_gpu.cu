#include "hip/hip_runtime.h"
// This file is part of Eigen, a lightweight C++ template library
// for linear algebra.
//
// Copyright (C) 2014 Benoit Steiner <benoit.steiner.goog@gmail.com>
//
// This Source Code Form is subject to the terms of the Mozilla
// Public License v. 2.0. If a copy of the MPL was not distributed
// with this file, You can obtain one at http://mozilla.org/MPL/2.0/.

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX

#define EIGEN_USE_GPU

#include "main.h"
#include <unsupported/Eigen/CXX11/Tensor>

#include <unsupported/Eigen/CXX11/src/Tensor/TensorGpuHipCudaDefines.h>

#define EIGEN_GPU_TEST_C99_MATH  EIGEN_HAS_CXX11

using Eigen::Tensor;

void test_gpu_nullary() {
  Tensor<float, 1, 0, int> in1(2);
  Tensor<float, 1, 0, int> in2(2);
  in1.setRandom();
  in2.setRandom();

  std::size_t tensor_bytes = in1.size() * sizeof(float);

  float* d_in1;
  float* d_in2;
  gpuMalloc((void**)(&d_in1), tensor_bytes);
  gpuMalloc((void**)(&d_in2), tensor_bytes);
  gpuMemcpy(d_in1, in1.data(), tensor_bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_in2, in2.data(), tensor_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 1, 0, int>, Eigen::Aligned> gpu_in1(
      d_in1, 2);
  Eigen::TensorMap<Eigen::Tensor<float, 1, 0, int>, Eigen::Aligned> gpu_in2(
      d_in2, 2);

  gpu_in1.device(gpu_device) = gpu_in1.constant(3.14f);
  gpu_in2.device(gpu_device) = gpu_in2.random();

  Tensor<float, 1, 0, int> new1(2);
  Tensor<float, 1, 0, int> new2(2);

  assert(gpuMemcpyAsync(new1.data(), d_in1, tensor_bytes, gpuMemcpyDeviceToHost,
                         gpu_device.stream()) == gpuSuccess);
  assert(gpuMemcpyAsync(new2.data(), d_in2, tensor_bytes, gpuMemcpyDeviceToHost,
                         gpu_device.stream()) == gpuSuccess);

  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 2; ++i) {
    VERIFY_IS_APPROX(new1(i), 3.14f);
    VERIFY_IS_NOT_EQUAL(new2(i), in2(i));
  }

  gpuFree(d_in1);
  gpuFree(d_in2);
}

void test_gpu_elementwise_small() {
  Tensor<float, 1> in1(Eigen::array<Eigen::DenseIndex, 1>(2));
  Tensor<float, 1> in2(Eigen::array<Eigen::DenseIndex, 1>(2));
  Tensor<float, 1> out(Eigen::array<Eigen::DenseIndex, 1>(2));
  in1.setRandom();
  in2.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t in2_bytes = in2.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_in1;
  float* d_in2;
  float* d_out;
  gpuMalloc((void**)(&d_in1), in1_bytes);
  gpuMalloc((void**)(&d_in2), in2_bytes);
  gpuMalloc((void**)(&d_out), out_bytes);

  gpuMemcpy(d_in1, in1.data(), in1_bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_in2, in2.data(), in2_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_in1(
      d_in1, Eigen::array<Eigen::DenseIndex, 1>(2));
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_in2(
      d_in2, Eigen::array<Eigen::DenseIndex, 1>(2));
  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_out(
      d_out, Eigen::array<Eigen::DenseIndex, 1>(2));

  gpu_out.device(gpu_device) = gpu_in1 + gpu_in2;

  assert(gpuMemcpyAsync(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost,
                         gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 2; ++i) {
    VERIFY_IS_APPROX(
        out(Eigen::array<Eigen::DenseIndex, 1>(i)),
        in1(Eigen::array<Eigen::DenseIndex, 1>(i)) + in2(Eigen::array<Eigen::DenseIndex, 1>(i)));
  }

  gpuFree(d_in1);
  gpuFree(d_in2);
  gpuFree(d_out);
}

void test_gpu_elementwise()
{
  Tensor<float, 3> in1(Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Tensor<float, 3> in2(Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Tensor<float, 3> in3(Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Tensor<float, 3> out(Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  in1.setRandom();
  in2.setRandom();
  in3.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t in2_bytes = in2.size() * sizeof(float);
  std::size_t in3_bytes = in3.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_in1;
  float* d_in2;
  float* d_in3;
  float* d_out;
  gpuMalloc((void**)(&d_in1), in1_bytes);
  gpuMalloc((void**)(&d_in2), in2_bytes);
  gpuMalloc((void**)(&d_in3), in3_bytes);
  gpuMalloc((void**)(&d_out), out_bytes);

  gpuMemcpy(d_in1, in1.data(), in1_bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_in2, in2.data(), in2_bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_in3, in3.data(), in3_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in1(d_in1, Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in2(d_in2, Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_in3(d_in3, Eigen::array<Eigen::DenseIndex, 3>(72,53,97));
  Eigen::TensorMap<Eigen::Tensor<float, 3> > gpu_out(d_out, Eigen::array<Eigen::DenseIndex, 3>(72,53,97));

  gpu_out.device(gpu_device) = gpu_in1 + gpu_in2 * gpu_in3;

  assert(gpuMemcpyAsync(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 53; ++j) {
      for (int k = 0; k < 97; ++k) {
        VERIFY_IS_APPROX(out(Eigen::array<Eigen::DenseIndex, 3>(i,j,k)), in1(Eigen::array<Eigen::DenseIndex, 3>(i,j,k)) + in2(Eigen::array<Eigen::DenseIndex, 3>(i,j,k)) * in3(Eigen::array<Eigen::DenseIndex, 3>(i,j,k)));
      }
    }
  }

  gpuFree(d_in1);
  gpuFree(d_in2);
  gpuFree(d_in3);
  gpuFree(d_out);
}

void test_gpu_props() {
  Tensor<float, 1> in1(200);
  Tensor<bool, 1> out(200);
  in1.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(bool);

  float* d_in1;
  bool* d_out;
  gpuMalloc((void**)(&d_in1), in1_bytes);
  gpuMalloc((void**)(&d_out), out_bytes);

  gpuMemcpy(d_in1, in1.data(), in1_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 1>, Eigen::Aligned> gpu_in1(
      d_in1, 200);
  Eigen::TensorMap<Eigen::Tensor<bool, 1>, Eigen::Aligned> gpu_out(
      d_out, 200);

  gpu_out.device(gpu_device) = (gpu_in1.isnan)();

  assert(gpuMemcpyAsync(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost,
                         gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 200; ++i) {
    VERIFY_IS_EQUAL(out(i), (std::isnan)(in1(i)));
  }

  gpuFree(d_in1);
  gpuFree(d_out);
}

void test_gpu_reduction()
{
  Tensor<float, 4> in1(72,53,97,113);
  Tensor<float, 2> out(72,97);
  in1.setRandom();

  std::size_t in1_bytes = in1.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_in1;
  float* d_out;
  gpuMalloc((void**)(&d_in1), in1_bytes);
  gpuMalloc((void**)(&d_out), out_bytes);

  gpuMemcpy(d_in1, in1.data(), in1_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4> > gpu_in1(d_in1, 72,53,97,113);
  Eigen::TensorMap<Eigen::Tensor<float, 2> > gpu_out(d_out, 72,97);

  array<Eigen::DenseIndex, 2> reduction_axis;
  reduction_axis[0] = 1;
  reduction_axis[1] = 3;

  gpu_out.device(gpu_device) = gpu_in1.maximum(reduction_axis);

  assert(gpuMemcpyAsync(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 97; ++j) {
      float expected = 0;
      for (int k = 0; k < 53; ++k) {
        for (int l = 0; l < 113; ++l) {
          expected =
              std::max<float>(expected, in1(i, k, j, l));
        }
      }
      VERIFY_IS_APPROX(out(i,j), expected);
    }
  }

  gpuFree(d_in1);
  gpuFree(d_out);
}

template<int DataLayout>
void test_gpu_contraction()
{
  // with these dimensions, the output has 300 * 140 elements, which is
  // more than 30 * 1024, which is the number of threads in blocks on
  // a 15 SM GK110 GPU
  Tensor<float, 4, DataLayout> t_left(6, 50, 3, 31);
  Tensor<float, 5, DataLayout> t_right(Eigen::array<Eigen::DenseIndex, 5>(3, 31, 7, 20, 1));
  Tensor<float, 5, DataLayout> t_result(Eigen::array<Eigen::DenseIndex, 5>(6, 50, 7, 20, 1));

  t_left.setRandom();
  t_right.setRandom();

  std::size_t t_left_bytes = t_left.size()  * sizeof(float);
  std::size_t t_right_bytes = t_right.size() * sizeof(float);
  std::size_t t_result_bytes = t_result.size() * sizeof(float);

  float* d_t_left;
  float* d_t_right;
  float* d_t_result;

  gpuMalloc((void**)(&d_t_left), t_left_bytes);
  gpuMalloc((void**)(&d_t_right), t_right_bytes);
  gpuMalloc((void**)(&d_t_result), t_result_bytes);

  gpuMemcpy(d_t_left, t_left.data(), t_left_bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_t_right, t_right.data(), t_right_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout> > gpu_t_left(d_t_left, 6, 50, 3, 31);
  Eigen::TensorMap<Eigen::Tensor<float, 5, DataLayout> > gpu_t_right(d_t_right, 3, 31, 7, 20, 1);
  Eigen::TensorMap<Eigen::Tensor<float, 5, DataLayout> > gpu_t_result(d_t_result, 6, 50, 7, 20, 1);

  typedef Eigen::Map<Eigen::Matrix<float, Dynamic, Dynamic, DataLayout> > MapXf;
  MapXf m_left(t_left.data(), 300, 93);
  MapXf m_right(t_right.data(), 93, 140);
  Eigen::Matrix<float, Dynamic, Dynamic, DataLayout> m_result(300, 140);

  typedef Tensor<float, 1>::DimensionPair DimPair;
  Eigen::array<DimPair, 2> dims;
  dims[0] = DimPair(2, 0);
  dims[1] = DimPair(3, 1);

  m_result = m_left * m_right;
  gpu_t_result.device(gpu_device) = gpu_t_left.contract(gpu_t_right, dims);

  gpuMemcpy(t_result.data(), d_t_result, t_result_bytes, gpuMemcpyDeviceToHost);

  for (DenseIndex i = 0; i < t_result.size(); i++) {
    if (fabs(t_result.data()[i] - m_result.data()[i]) >= 1e-4f) {
      std::cout << "mismatch detected at index " << i << ": " << t_result.data()[i] << " vs " <<  m_result.data()[i] << std::endl;
      assert(false);
    }
  }

  gpuFree(d_t_left);
  gpuFree(d_t_right);
  gpuFree(d_t_result);
}

template<int DataLayout>
void test_gpu_convolution_1d()
{
  Tensor<float, 4, DataLayout> input(74,37,11,137);
  Tensor<float, 1, DataLayout> kernel(4);
  Tensor<float, 4, DataLayout> out(74,34,11,137);
  input = input.constant(10.0f) + input.random();
  kernel = kernel.constant(7.0f) + kernel.random();

  std::size_t input_bytes = input.size() * sizeof(float);
  std::size_t kernel_bytes = kernel.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_input;
  float* d_kernel;
  float* d_out;
  gpuMalloc((void**)(&d_input), input_bytes);
  gpuMalloc((void**)(&d_kernel), kernel_bytes);
  gpuMalloc((void**)(&d_out), out_bytes);

  gpuMemcpy(d_input, input.data(), input_bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_kernel, kernel.data(), kernel_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout> > gpu_input(d_input, 74,37,11,137);
  Eigen::TensorMap<Eigen::Tensor<float, 1, DataLayout> > gpu_kernel(d_kernel, 4);
  Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout> > gpu_out(d_out, 74,34,11,137);

  Eigen::array<Eigen::DenseIndex, 1> dims(1);
  gpu_out.device(gpu_device) = gpu_input.convolve(gpu_kernel, dims);

  assert(gpuMemcpyAsync(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 74; ++i) {
    for (int j = 0; j < 34; ++j) {
      for (int k = 0; k < 11; ++k) {
        for (int l = 0; l < 137; ++l) {
          const float result = out(i,j,k,l);
          const float expected = input(i,j+0,k,l) * kernel(0) + input(i,j+1,k,l) * kernel(1) +
                                 input(i,j+2,k,l) * kernel(2) + input(i,j+3,k,l) * kernel(3);
          VERIFY_IS_APPROX(result, expected);
        }
      }
    }
  }

  gpuFree(d_input);
  gpuFree(d_kernel);
  gpuFree(d_out);
}

void test_gpu_convolution_inner_dim_col_major_1d()
{
  Tensor<float, 4, ColMajor> input(74,9,11,7);
  Tensor<float, 1, ColMajor> kernel(4);
  Tensor<float, 4, ColMajor> out(71,9,11,7);
  input = input.constant(10.0f) + input.random();
  kernel = kernel.constant(7.0f) + kernel.random();

  std::size_t input_bytes = input.size() * sizeof(float);
  std::size_t kernel_bytes = kernel.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_input;
  float* d_kernel;
  float* d_out;
  gpuMalloc((void**)(&d_input), input_bytes);
  gpuMalloc((void**)(&d_kernel), kernel_bytes);
  gpuMalloc((void**)(&d_out), out_bytes);

  gpuMemcpy(d_input, input.data(), input_bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_kernel, kernel.data(), kernel_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4, ColMajor> > gpu_input(d_input,74,9,11,7);
  Eigen::TensorMap<Eigen::Tensor<float, 1, ColMajor> > gpu_kernel(d_kernel,4);
  Eigen::TensorMap<Eigen::Tensor<float, 4, ColMajor> > gpu_out(d_out,71,9,11,7);

  Eigen::array<Eigen::DenseIndex, 1> dims(0);
  gpu_out.device(gpu_device) = gpu_input.convolve(gpu_kernel, dims);

  assert(gpuMemcpyAsync(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 71; ++i) {
    for (int j = 0; j < 9; ++j) {
      for (int k = 0; k < 11; ++k) {
        for (int l = 0; l < 7; ++l) {
          const float result = out(i,j,k,l);
          const float expected = input(i+0,j,k,l) * kernel(0) + input(i+1,j,k,l) * kernel(1) +
                                 input(i+2,j,k,l) * kernel(2) + input(i+3,j,k,l) * kernel(3);
          VERIFY_IS_APPROX(result, expected);
        }
      }
    }
  }

  gpuFree(d_input);
  gpuFree(d_kernel);
  gpuFree(d_out);
}

void test_gpu_convolution_inner_dim_row_major_1d()
{
  Tensor<float, 4, RowMajor> input(7,9,11,74);
  Tensor<float, 1, RowMajor> kernel(4);
  Tensor<float, 4, RowMajor> out(7,9,11,71);
  input = input.constant(10.0f) + input.random();
  kernel = kernel.constant(7.0f) + kernel.random();

  std::size_t input_bytes = input.size() * sizeof(float);
  std::size_t kernel_bytes = kernel.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_input;
  float* d_kernel;
  float* d_out;
  gpuMalloc((void**)(&d_input), input_bytes);
  gpuMalloc((void**)(&d_kernel), kernel_bytes);
  gpuMalloc((void**)(&d_out), out_bytes);

  gpuMemcpy(d_input, input.data(), input_bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_kernel, kernel.data(), kernel_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4, RowMajor> > gpu_input(d_input, 7,9,11,74);
  Eigen::TensorMap<Eigen::Tensor<float, 1, RowMajor> > gpu_kernel(d_kernel, 4);
  Eigen::TensorMap<Eigen::Tensor<float, 4, RowMajor> > gpu_out(d_out, 7,9,11,71);

  Eigen::array<Eigen::DenseIndex, 1> dims(3);
  gpu_out.device(gpu_device) = gpu_input.convolve(gpu_kernel, dims);

  assert(gpuMemcpyAsync(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 7; ++i) {
    for (int j = 0; j < 9; ++j) {
      for (int k = 0; k < 11; ++k) {
        for (int l = 0; l < 71; ++l) {
          const float result = out(i,j,k,l);
          const float expected = input(i,j,k,l+0) * kernel(0) + input(i,j,k,l+1) * kernel(1) +
                                 input(i,j,k,l+2) * kernel(2) + input(i,j,k,l+3) * kernel(3);
          VERIFY_IS_APPROX(result, expected);
        }
      }
    }
  }

  gpuFree(d_input);
  gpuFree(d_kernel);
  gpuFree(d_out);
}

template<int DataLayout>
void test_gpu_convolution_2d()
{
  Tensor<float, 4, DataLayout> input(74,37,11,137);
  Tensor<float, 2, DataLayout> kernel(3,4);
  Tensor<float, 4, DataLayout> out(74,35,8,137);
  input = input.constant(10.0f) + input.random();
  kernel = kernel.constant(7.0f) + kernel.random();

  std::size_t input_bytes = input.size() * sizeof(float);
  std::size_t kernel_bytes = kernel.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_input;
  float* d_kernel;
  float* d_out;
  gpuMalloc((void**)(&d_input), input_bytes);
  gpuMalloc((void**)(&d_kernel), kernel_bytes);
  gpuMalloc((void**)(&d_out), out_bytes);

  gpuMemcpy(d_input, input.data(), input_bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_kernel, kernel.data(), kernel_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout> > gpu_input(d_input,74,37,11,137);
  Eigen::TensorMap<Eigen::Tensor<float, 2, DataLayout> > gpu_kernel(d_kernel,3,4);
  Eigen::TensorMap<Eigen::Tensor<float, 4, DataLayout> > gpu_out(d_out,74,35,8,137);

  Eigen::array<Eigen::DenseIndex, 2> dims(1,2);
  gpu_out.device(gpu_device) = gpu_input.convolve(gpu_kernel, dims);

  assert(gpuMemcpyAsync(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 74; ++i) {
    for (int j = 0; j < 35; ++j) {
      for (int k = 0; k < 8; ++k) {
        for (int l = 0; l < 137; ++l) {
          const float result = out(i,j,k,l);
          const float expected = input(i,j+0,k+0,l) * kernel(0,0) +
                                 input(i,j+1,k+0,l) * kernel(1,0) +
                                 input(i,j+2,k+0,l) * kernel(2,0) +
                                 input(i,j+0,k+1,l) * kernel(0,1) +
                                 input(i,j+1,k+1,l) * kernel(1,1) +
                                 input(i,j+2,k+1,l) * kernel(2,1) +
                                 input(i,j+0,k+2,l) * kernel(0,2) +
                                 input(i,j+1,k+2,l) * kernel(1,2) +
                                 input(i,j+2,k+2,l) * kernel(2,2) +
                                 input(i,j+0,k+3,l) * kernel(0,3) +
                                 input(i,j+1,k+3,l) * kernel(1,3) +
                                 input(i,j+2,k+3,l) * kernel(2,3);
          VERIFY_IS_APPROX(result, expected);
        }
      }
    }
  }

  gpuFree(d_input);
  gpuFree(d_kernel);
  gpuFree(d_out);
}

template<int DataLayout>
void test_gpu_convolution_3d()
{
  Tensor<float, 5, DataLayout> input(Eigen::array<Eigen::DenseIndex, 5>(74,37,11,137,17));
  Tensor<float, 3, DataLayout> kernel(3,4,2);
  Tensor<float, 5, DataLayout> out(Eigen::array<Eigen::DenseIndex, 5>(74,35,8,136,17));
  input = input.constant(10.0f) + input.random();
  kernel = kernel.constant(7.0f) + kernel.random();

  std::size_t input_bytes = input.size() * sizeof(float);
  std::size_t kernel_bytes = kernel.size() * sizeof(float);
  std::size_t out_bytes = out.size() * sizeof(float);

  float* d_input;
  float* d_kernel;
  float* d_out;
  gpuMalloc((void**)(&d_input), input_bytes);
  gpuMalloc((void**)(&d_kernel), kernel_bytes);
  gpuMalloc((void**)(&d_out), out_bytes);

  gpuMemcpy(d_input, input.data(), input_bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_kernel, kernel.data(), kernel_bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;    
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<float, 5, DataLayout> > gpu_input(d_input,74,37,11,137,17);
  Eigen::TensorMap<Eigen::Tensor<float, 3, DataLayout> > gpu_kernel(d_kernel,3,4,2);
  Eigen::TensorMap<Eigen::Tensor<float, 5, DataLayout> > gpu_out(d_out,74,35,8,136,17);

  Eigen::array<Eigen::DenseIndex, 3> dims(1,2,3);
  gpu_out.device(gpu_device) = gpu_input.convolve(gpu_kernel, dims);

  assert(gpuMemcpyAsync(out.data(), d_out, out_bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 74; ++i) {
    for (int j = 0; j < 35; ++j) {
      for (int k = 0; k < 8; ++k) {
        for (int l = 0; l < 136; ++l) {
          for (int m = 0; m < 17; ++m) {
            const float result = out(i,j,k,l,m);
            const float expected = input(i,j+0,k+0,l+0,m) * kernel(0,0,0) +
                                   input(i,j+1,k+0,l+0,m) * kernel(1,0,0) +
                                   input(i,j+2,k+0,l+0,m) * kernel(2,0,0) +
                                   input(i,j+0,k+1,l+0,m) * kernel(0,1,0) +
                                   input(i,j+1,k+1,l+0,m) * kernel(1,1,0) +
                                   input(i,j+2,k+1,l+0,m) * kernel(2,1,0) +
                                   input(i,j+0,k+2,l+0,m) * kernel(0,2,0) +
                                   input(i,j+1,k+2,l+0,m) * kernel(1,2,0) +
                                   input(i,j+2,k+2,l+0,m) * kernel(2,2,0) +
                                   input(i,j+0,k+3,l+0,m) * kernel(0,3,0) +
                                   input(i,j+1,k+3,l+0,m) * kernel(1,3,0) +
                                   input(i,j+2,k+3,l+0,m) * kernel(2,3,0) +
                                   input(i,j+0,k+0,l+1,m) * kernel(0,0,1) +
                                   input(i,j+1,k+0,l+1,m) * kernel(1,0,1) +
                                   input(i,j+2,k+0,l+1,m) * kernel(2,0,1) +
                                   input(i,j+0,k+1,l+1,m) * kernel(0,1,1) +
                                   input(i,j+1,k+1,l+1,m) * kernel(1,1,1) +
                                   input(i,j+2,k+1,l+1,m) * kernel(2,1,1) +
                                   input(i,j+0,k+2,l+1,m) * kernel(0,2,1) +
                                   input(i,j+1,k+2,l+1,m) * kernel(1,2,1) +
                                   input(i,j+2,k+2,l+1,m) * kernel(2,2,1) +
                                   input(i,j+0,k+3,l+1,m) * kernel(0,3,1) +
                                   input(i,j+1,k+3,l+1,m) * kernel(1,3,1) +
                                   input(i,j+2,k+3,l+1,m) * kernel(2,3,1);
            VERIFY_IS_APPROX(result, expected);
          }
        }
      }
    }
  }

  gpuFree(d_input);
  gpuFree(d_kernel);
  gpuFree(d_out);
}


#if EIGEN_GPU_TEST_C99_MATH
template <typename Scalar>
void test_gpu_lgamma(const Scalar stddev)
{
  Tensor<Scalar, 2> in(72,97);
  in.setRandom();
  in *= in.constant(stddev);
  Tensor<Scalar, 2> out(72,97);
  out.setZero();

  std::size_t bytes = in.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out;
  gpuMalloc((void**)(&d_in), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_in, in.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_in(d_in, 72, 97);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_out(d_out, 72, 97);

  gpu_out.device(gpu_device) = gpu_in.lgamma();

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 97; ++j) {
      VERIFY_IS_APPROX(out(i,j), (std::lgamma)(in(i,j)));
    }
  }

  gpuFree(d_in);
  gpuFree(d_out);
}
#endif

template <typename Scalar>
void test_gpu_digamma()
{
  Tensor<Scalar, 1> in(7);
  Tensor<Scalar, 1> out(7);
  Tensor<Scalar, 1> expected_out(7);
  out.setZero();

  in(0) = Scalar(1);
  in(1) = Scalar(1.5);
  in(2) = Scalar(4);
  in(3) = Scalar(-10.5);
  in(4) = Scalar(10000.5);
  in(5) = Scalar(0);
  in(6) = Scalar(-1);

  expected_out(0) = Scalar(-0.5772156649015329);
  expected_out(1) = Scalar(0.03648997397857645);
  expected_out(2) = Scalar(1.2561176684318);
  expected_out(3) = Scalar(2.398239129535781);
  expected_out(4) = Scalar(9.210340372392849);
  expected_out(5) = std::numeric_limits<Scalar>::infinity();
  expected_out(6) = std::numeric_limits<Scalar>::infinity();

  std::size_t bytes = in.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out;
  gpuMalloc((void**)(&d_in), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_in, in.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in(d_in, 7);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 7);

  gpu_out.device(gpu_device) = gpu_in.digamma();

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 5; ++i) {
    VERIFY_IS_APPROX(out(i), expected_out(i));
  }
  for (int i = 5; i < 7; ++i) {
    VERIFY_IS_EQUAL(out(i), expected_out(i));
  }

  gpuFree(d_in);
  gpuFree(d_out);
}

template <typename Scalar>
void test_gpu_zeta()
{
  Tensor<Scalar, 1> in_x(6);
  Tensor<Scalar, 1> in_q(6);
  Tensor<Scalar, 1> out(6);
  Tensor<Scalar, 1> expected_out(6);
  out.setZero();

  in_x(0) = Scalar(1);
  in_x(1) = Scalar(1.5);
  in_x(2) = Scalar(4);
  in_x(3) = Scalar(-10.5);
  in_x(4) = Scalar(10000.5);
  in_x(5) = Scalar(3);
  
  in_q(0) = Scalar(1.2345);
  in_q(1) = Scalar(2);
  in_q(2) = Scalar(1.5);
  in_q(3) = Scalar(3);
  in_q(4) = Scalar(1.0001);
  in_q(5) = Scalar(-2.5);

  expected_out(0) = std::numeric_limits<Scalar>::infinity();
  expected_out(1) = Scalar(1.61237534869);
  expected_out(2) = Scalar(0.234848505667);
  expected_out(3) = Scalar(1.03086757337e-5);
  expected_out(4) = Scalar(0.367879440865);
  expected_out(5) = Scalar(0.054102025820864097);

  std::size_t bytes = in_x.size() * sizeof(Scalar);

  Scalar* d_in_x;
  Scalar* d_in_q;
  Scalar* d_out;
  gpuMalloc((void**)(&d_in_x), bytes);
  gpuMalloc((void**)(&d_in_q), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_in_x, in_x.data(), bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_in_q, in_q.data(), bytes, gpuMemcpyHostToDevice);
  
  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in_x(d_in_x, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in_q(d_in_q, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 6);

  gpu_out.device(gpu_device) = gpu_in_x.zeta(gpu_in_q);

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  VERIFY_IS_EQUAL(out(0), expected_out(0));
  VERIFY((std::isnan)(out(3)));

  for (int i = 1; i < 6; ++i) {
    if (i != 3) {
      VERIFY_IS_APPROX(out(i), expected_out(i));
    }
  }

  gpuFree(d_in_x);
  gpuFree(d_in_q);
  gpuFree(d_out);
}

template <typename Scalar>
void test_gpu_polygamma()
{
  Tensor<Scalar, 1> in_x(7);
  Tensor<Scalar, 1> in_n(7);
  Tensor<Scalar, 1> out(7);
  Tensor<Scalar, 1> expected_out(7);
  out.setZero();

  in_n(0) = Scalar(1);
  in_n(1) = Scalar(1);
  in_n(2) = Scalar(1);
  in_n(3) = Scalar(17);
  in_n(4) = Scalar(31);
  in_n(5) = Scalar(28);
  in_n(6) = Scalar(8);
  
  in_x(0) = Scalar(2);
  in_x(1) = Scalar(3);
  in_x(2) = Scalar(25.5);
  in_x(3) = Scalar(4.7);
  in_x(4) = Scalar(11.8);
  in_x(5) = Scalar(17.7);
  in_x(6) = Scalar(30.2);

  expected_out(0) = Scalar(0.644934066848);
  expected_out(1) = Scalar(0.394934066848);
  expected_out(2) = Scalar(0.0399946696496);
  expected_out(3) = Scalar(293.334565435);
  expected_out(4) = Scalar(0.445487887616);
  expected_out(5) = Scalar(-2.47810300902e-07);
  expected_out(6) = Scalar(-8.29668781082e-09);

  std::size_t bytes = in_x.size() * sizeof(Scalar);

  Scalar* d_in_x;
  Scalar* d_in_n;
  Scalar* d_out;
  gpuMalloc((void**)(&d_in_x), bytes);
  gpuMalloc((void**)(&d_in_n), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_in_x, in_x.data(), bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_in_n, in_n.data(), bytes, gpuMemcpyHostToDevice);
  
  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in_x(d_in_x, 7);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in_n(d_in_n, 7);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 7);

  gpu_out.device(gpu_device) = gpu_in_n.polygamma(gpu_in_x);

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 7; ++i) {
    VERIFY_IS_APPROX(out(i), expected_out(i));
  }

  gpuFree(d_in_x);
  gpuFree(d_in_n);
  gpuFree(d_out);
}

template <typename Scalar>
void test_gpu_igamma()
{
  Tensor<Scalar, 2> a(6, 6);
  Tensor<Scalar, 2> x(6, 6);
  Tensor<Scalar, 2> out(6, 6);
  out.setZero();

  Scalar a_s[] = {Scalar(0), Scalar(1), Scalar(1.5), Scalar(4), Scalar(0.0001), Scalar(1000.5)};
  Scalar x_s[] = {Scalar(0), Scalar(1), Scalar(1.5), Scalar(4), Scalar(0.0001), Scalar(1000.5)};

  for (int i = 0; i < 6; ++i) {
    for (int j = 0; j < 6; ++j) {
      a(i, j) = a_s[i];
      x(i, j) = x_s[j];
    }
  }

  Scalar nan = std::numeric_limits<Scalar>::quiet_NaN();
  Scalar igamma_s[][6] = {{0.0, nan, nan, nan, nan, nan},
                          {0.0, 0.6321205588285578, 0.7768698398515702,
                           0.9816843611112658, 9.999500016666262e-05, 1.0},
                          {0.0, 0.4275932955291202, 0.608374823728911,
                           0.9539882943107686, 7.522076445089201e-07, 1.0},
                          {0.0, 0.01898815687615381, 0.06564245437845008,
                           0.5665298796332909, 4.166333347221828e-18, 1.0},
                          {0.0, 0.9999780593618628, 0.9999899967080838,
                           0.9999996219837988, 0.9991370418689945, 1.0},
                          {0.0, 0.0, 0.0, 0.0, 0.0, 0.5042041932513908}};



  std::size_t bytes = a.size() * sizeof(Scalar);

  Scalar* d_a;
  Scalar* d_x;
  Scalar* d_out;
  assert(gpuMalloc((void**)(&d_a), bytes) == gpuSuccess);
  assert(gpuMalloc((void**)(&d_x), bytes) == gpuSuccess);
  assert(gpuMalloc((void**)(&d_out), bytes) == gpuSuccess);

  gpuMemcpy(d_a, a.data(), bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_x, x.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_a(d_a, 6, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_x(d_x, 6, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_out(d_out, 6, 6);

  gpu_out.device(gpu_device) = gpu_a.igamma(gpu_x);

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 6; ++i) {
    for (int j = 0; j < 6; ++j) {
      if ((std::isnan)(igamma_s[i][j])) {
        VERIFY((std::isnan)(out(i, j)));
      } else {
        VERIFY_IS_APPROX(out(i, j), igamma_s[i][j]);
      }
    }
  }

  gpuFree(d_a);
  gpuFree(d_x);
  gpuFree(d_out);
}

template <typename Scalar>
void test_gpu_igammac()
{
  Tensor<Scalar, 2> a(6, 6);
  Tensor<Scalar, 2> x(6, 6);
  Tensor<Scalar, 2> out(6, 6);
  out.setZero();

  Scalar a_s[] = {Scalar(0), Scalar(1), Scalar(1.5), Scalar(4), Scalar(0.0001), Scalar(1000.5)};
  Scalar x_s[] = {Scalar(0), Scalar(1), Scalar(1.5), Scalar(4), Scalar(0.0001), Scalar(1000.5)};

  for (int i = 0; i < 6; ++i) {
    for (int j = 0; j < 6; ++j) {
      a(i, j) = a_s[i];
      x(i, j) = x_s[j];
    }
  }

  Scalar nan = std::numeric_limits<Scalar>::quiet_NaN();
  Scalar igammac_s[][6] = {{nan, nan, nan, nan, nan, nan},
                           {1.0, 0.36787944117144233, 0.22313016014842982,
                            0.018315638888734182, 0.9999000049998333, 0.0},
                           {1.0, 0.5724067044708798, 0.3916251762710878,
                            0.04601170568923136, 0.9999992477923555, 0.0},
                           {1.0, 0.9810118431238462, 0.9343575456215499,
                            0.4334701203667089, 1.0, 0.0},
                           {1.0, 2.1940638138146658e-05, 1.0003291916285e-05,
                            3.7801620118431334e-07, 0.0008629581310054535,
                            0.0},
                           {1.0, 1.0, 1.0, 1.0, 1.0, 0.49579580674813944}};

  std::size_t bytes = a.size() * sizeof(Scalar);

  Scalar* d_a;
  Scalar* d_x;
  Scalar* d_out;
  gpuMalloc((void**)(&d_a), bytes);
  gpuMalloc((void**)(&d_x), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_a, a.data(), bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_x, x.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_a(d_a, 6, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_x(d_x, 6, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_out(d_out, 6, 6);

  gpu_out.device(gpu_device) = gpu_a.igammac(gpu_x);

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 6; ++i) {
    for (int j = 0; j < 6; ++j) {
      if ((std::isnan)(igammac_s[i][j])) {
        VERIFY((std::isnan)(out(i, j)));
      } else {
        VERIFY_IS_APPROX(out(i, j), igammac_s[i][j]);
      }
    }
  }

  gpuFree(d_a);
  gpuFree(d_x);
  gpuFree(d_out);
}

#if EIGEN_GPU_TEST_C99_MATH
template <typename Scalar>
void test_gpu_erf(const Scalar stddev)
{
  Tensor<Scalar, 2> in(72,97);
  in.setRandom();
  in *= in.constant(stddev);
  Tensor<Scalar, 2> out(72,97);
  out.setZero();

  std::size_t bytes = in.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out;
  assert(gpuMalloc((void**)(&d_in), bytes) == gpuSuccess);
  assert(gpuMalloc((void**)(&d_out), bytes) == gpuSuccess);

  gpuMemcpy(d_in, in.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_in(d_in, 72, 97);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_out(d_out, 72, 97);

  gpu_out.device(gpu_device) = gpu_in.erf();

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 97; ++j) {
      VERIFY_IS_APPROX(out(i,j), (std::erf)(in(i,j)));
    }
  }

  gpuFree(d_in);
  gpuFree(d_out);
}

template <typename Scalar>
void test_gpu_erfc(const Scalar stddev)
{
  Tensor<Scalar, 2> in(72,97);
  in.setRandom();
  in *= in.constant(stddev);
  Tensor<Scalar, 2> out(72,97);
  out.setZero();

  std::size_t bytes = in.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out;
  gpuMalloc((void**)(&d_in), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_in, in.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_in(d_in, 72, 97);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 2> > gpu_out(d_out, 72, 97);

  gpu_out.device(gpu_device) = gpu_in.erfc();

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 72; ++i) {
    for (int j = 0; j < 97; ++j) {
      VERIFY_IS_APPROX(out(i,j), (std::erfc)(in(i,j)));
    }
  }

  gpuFree(d_in);
  gpuFree(d_out);
}
#endif
template <typename Scalar>
void test_gpu_ndtri()
{
  Tensor<Scalar, 1> in_x(8);
  Tensor<Scalar, 1> out(8);
  Tensor<Scalar, 1> expected_out(8);
  out.setZero();

  in_x(0) = Scalar(1);
  in_x(1) = Scalar(0.);
  in_x(2) = Scalar(0.5);
  in_x(3) = Scalar(0.2);
  in_x(4) = Scalar(0.8);
  in_x(5) = Scalar(0.9);
  in_x(6) = Scalar(0.1);
  in_x(7) = Scalar(0.99);
  in_x(8) = Scalar(0.01);

  expected_out(0) = std::numeric_limits<Scalar>::infinity();
  expected_out(1) = -std::numeric_limits<Scalar>::infinity();
  expected_out(2) = Scalar(0.0);
  expected_out(3) = Scalar(-0.8416212335729142);
  expected_out(4) = Scalar(0.8416212335729142);
  expected_out(5) = Scalar(1.2815515655446004);
  expected_out(6) = Scalar(-1.2815515655446004);
  expected_out(7) = Scalar(2.3263478740408408);
  expected_out(8) = Scalar(-2.3263478740408408);

  std::size_t bytes = in_x.size() * sizeof(Scalar);

  Scalar* d_in_x;
  Scalar* d_out;
  gpuMalloc((void**)(&d_in_x), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_in_x, in_x.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in_x(d_in_x, 6);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 6);

  gpu_out.device(gpu_device) = gpu_in_x.ndtri();

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  VERIFY_IS_EQUAL(out(0), expected_out(0));
  VERIFY((std::isnan)(out(3)));

  for (int i = 1; i < 6; ++i) {
    if (i != 3) {
      VERIFY_IS_APPROX(out(i), expected_out(i));
    }
  }

  gpuFree(d_in_x);
  gpuFree(d_out);
}

template <typename Scalar>
void test_gpu_betainc()
{
  Tensor<Scalar, 1> in_x(125);
  Tensor<Scalar, 1> in_a(125);
  Tensor<Scalar, 1> in_b(125);
  Tensor<Scalar, 1> out(125);
  Tensor<Scalar, 1> expected_out(125);
  out.setZero();

  Scalar nan = std::numeric_limits<Scalar>::quiet_NaN();

  Array<Scalar, 1, Dynamic> x(125);
  Array<Scalar, 1, Dynamic> a(125);
  Array<Scalar, 1, Dynamic> b(125);
  Array<Scalar, 1, Dynamic> v(125);

  a << 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0,
      0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999,
      0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999,
      0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 0.999, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379, 999.999, 999.999,
      999.999, 999.999, 999.999, 999.999, 999.999, 999.999, 999.999, 999.999,
      999.999, 999.999, 999.999, 999.999, 999.999, 999.999, 999.999, 999.999,
      999.999, 999.999, 999.999, 999.999, 999.999, 999.999, 999.999;

  b << 0.0, 0.0, 0.0, 0.0, 0.0, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379, 0.999,
      0.999, 0.999, 0.999, 0.999, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379, 999.999, 999.999,
      999.999, 999.999, 999.999, 0.0, 0.0, 0.0, 0.0, 0.0, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.999, 0.999, 0.999, 0.999, 0.999, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 999.999, 999.999, 999.999, 999.999, 999.999, 0.0, 0.0,
      0.0, 0.0, 0.0, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379, 0.999,
      0.999, 0.999, 0.999, 0.999, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379, 999.999, 999.999,
      999.999, 999.999, 999.999, 0.0, 0.0, 0.0, 0.0, 0.0, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.999, 0.999, 0.999, 0.999, 0.999, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 999.999, 999.999, 999.999, 999.999, 999.999, 0.0, 0.0,
      0.0, 0.0, 0.0, 0.03062277660168379, 0.03062277660168379,
      0.03062277660168379, 0.03062277660168379, 0.03062277660168379, 0.999,
      0.999, 0.999, 0.999, 0.999, 31.62177660168379, 31.62177660168379,
      31.62177660168379, 31.62177660168379, 31.62177660168379, 999.999, 999.999,
      999.999, 999.999, 999.999;

  x << -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8,
      1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5,
      0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2,
      0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1,
      0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1,
      -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8,
      1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5,
      0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2,
      0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1, -0.1, 0.2, 0.5, 0.8, 1.1;

  v << nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan,
      nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan, nan,
      nan, nan, 0.47972119876364683, 0.5, 0.5202788012363533, nan, nan,
      0.9518683957740043, 0.9789663010413743, 0.9931729188073435, nan, nan,
      0.999995949033062, 0.9999999999993698, 0.9999999999999999, nan, nan,
      0.9999999999999999, 0.9999999999999999, 0.9999999999999999, nan, nan, nan,
      nan, nan, nan, nan, 0.006827081192655869, 0.0210336989586256,
      0.04813160422599567, nan, nan, 0.20014344256217678, 0.5000000000000001,
      0.7998565574378232, nan, nan, 0.9991401428435834, 0.999999999698403,
      0.9999999999999999, nan, nan, 0.9999999999999999, 0.9999999999999999,
      0.9999999999999999, nan, nan, nan, nan, nan, nan, nan,
      1.0646600232370887e-25, 6.301722877826246e-13, 4.050966937974938e-06, nan,
      nan, 7.864342668429763e-23, 3.015969667594166e-10, 0.0008598571564165444,
      nan, nan, 6.031987710123844e-08, 0.5000000000000007, 0.9999999396801229,
      nan, nan, 0.9999999999999999, 0.9999999999999999, 0.9999999999999999, nan,
      nan, nan, nan, nan, nan, nan, 0.0, 7.029920380986636e-306,
      2.2450728208591345e-101, nan, nan, 0.0, 9.275871147869727e-302,
      1.2232913026152827e-97, nan, nan, 0.0, 3.0891393081932924e-252,
      2.9303043666183996e-60, nan, nan, 2.248913486879199e-196,
      0.5000000000004947, 0.9999999999999999, nan;

  for (int i = 0; i < 125; ++i) {
    in_x(i) = x(i);
    in_a(i) = a(i);
    in_b(i) = b(i);
    expected_out(i) = v(i);
  }

  std::size_t bytes = in_x.size() * sizeof(Scalar);

  Scalar* d_in_x;
  Scalar* d_in_a;
  Scalar* d_in_b;
  Scalar* d_out;
  gpuMalloc((void**)(&d_in_x), bytes);
  gpuMalloc((void**)(&d_in_a), bytes);
  gpuMalloc((void**)(&d_in_b), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_in_x, in_x.data(), bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_in_a, in_a.data(), bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_in_b, in_b.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in_x(d_in_x, 125);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in_a(d_in_a, 125);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in_b(d_in_b, 125);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 125);

  gpu_out.device(gpu_device) = betainc(gpu_in_a, gpu_in_b, gpu_in_x);

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost, gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 1; i < 125; ++i) {
    if ((std::isnan)(expected_out(i))) {
      VERIFY((std::isnan)(out(i)));
    } else {
      VERIFY_IS_APPROX(out(i), expected_out(i));
    }
  }

  gpuFree(d_in_x);
  gpuFree(d_in_a);
  gpuFree(d_in_b);
  gpuFree(d_out);
}

template <typename Scalar>
void test_gpu_i0e()
{
  Tensor<Scalar, 1> in_x(21);
  Tensor<Scalar, 1> out(21);
  Tensor<Scalar, 1> expected_out(21);
  out.setZero();

  Array<Scalar, 1, Dynamic> in_x_array(21);
  Array<Scalar, 1, Dynamic> expected_out_array(21);

  in_x_array << -20.0, -18.0, -16.0, -14.0, -12.0, -10.0, -8.0, -6.0, -4.0,
      -2.0, 0.0, 2.0, 4.0, 6.0, 8.0, 10.0, 12.0, 14.0, 16.0, 18.0, 20.0;

  expected_out_array << 0.0897803118848, 0.0947062952128, 0.100544127361,
      0.107615251671, 0.116426221213, 0.127833337163, 0.143431781857,
      0.16665743264, 0.207001921224, 0.308508322554, 1.0, 0.308508322554,
      0.207001921224, 0.16665743264, 0.143431781857, 0.127833337163,
      0.116426221213, 0.107615251671, 0.100544127361, 0.0947062952128,
      0.0897803118848;

  for (int i = 0; i < 21; ++i) {
    in_x(i) = in_x_array(i);
    expected_out(i) = expected_out_array(i);
  }

  std::size_t bytes = in_x.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out;
  gpuMalloc((void**)(&d_in), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_in, in_x.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in(d_in, 21);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 21);

  gpu_out.device(gpu_device) = gpu_in.bessel_i0e();

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost,
                         gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 21; ++i) {
    VERIFY_IS_APPROX(out(i), expected_out(i));
  }

  gpuFree(d_in);
  gpuFree(d_out);
}

template <typename Scalar>
void test_gpu_i1e()
{
  Tensor<Scalar, 1> in_x(21);
  Tensor<Scalar, 1> out(21);
  Tensor<Scalar, 1> expected_out(21);
  out.setZero();

  Array<Scalar, 1, Dynamic> in_x_array(21);
  Array<Scalar, 1, Dynamic> expected_out_array(21);

  in_x_array << -20.0, -18.0, -16.0, -14.0, -12.0, -10.0, -8.0, -6.0, -4.0,
      -2.0, 0.0, 2.0, 4.0, 6.0, 8.0, 10.0, 12.0, 14.0, 16.0, 18.0, 20.0;

  expected_out_array << -0.0875062221833, -0.092036796872, -0.0973496147565,
      -0.103697667463, -0.11146429929, -0.121262681384, -0.134142493293,
      -0.152051459309, -0.178750839502, -0.215269289249, 0.0, 0.215269289249,
      0.178750839502, 0.152051459309, 0.134142493293, 0.121262681384,
      0.11146429929, 0.103697667463, 0.0973496147565, 0.092036796872,
      0.0875062221833;

  for (int i = 0; i < 21; ++i) {
    in_x(i) = in_x_array(i);
    expected_out(i) = expected_out_array(i);
  }

  std::size_t bytes = in_x.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out;
  gpuMalloc((void**)(&d_in), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_in, in_x.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in(d_in, 21);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 21);

  gpu_out.device(gpu_device) = gpu_in.bessel_i1e();

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost,
                         gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 21; ++i) {
    VERIFY_IS_APPROX(out(i), expected_out(i));
  }

  gpuFree(d_in);
  gpuFree(d_out);
}

template <typename Scalar>
void test_gpu_igamma_der_a()
{
  Tensor<Scalar, 1> in_x(30);
  Tensor<Scalar, 1> in_a(30);
  Tensor<Scalar, 1> out(30);
  Tensor<Scalar, 1> expected_out(30);
  out.setZero();

  Array<Scalar, 1, Dynamic> in_a_array(30);
  Array<Scalar, 1, Dynamic> in_x_array(30);
  Array<Scalar, 1, Dynamic> expected_out_array(30);

  // See special_functions.cpp for the Python code that generates the test data.

  in_a_array << 0.01, 0.01, 0.01, 0.01, 0.01, 0.1, 0.1, 0.1, 0.1, 0.1, 1.0, 1.0,
      1.0, 1.0, 1.0, 10.0, 10.0, 10.0, 10.0, 10.0, 100.0, 100.0, 100.0, 100.0,
      100.0, 1000.0, 1000.0, 1000.0, 1000.0, 1000.0;

  in_x_array << 1.25668890405e-26, 1.17549435082e-38, 1.20938905072e-05,
      1.17549435082e-38, 1.17549435082e-38, 5.66572070696e-16, 0.0132865061065,
      0.0200034203853, 6.29263709118e-17, 1.37160367764e-06, 0.333412038288,
      1.18135687766, 0.580629033777, 0.170631439426, 0.786686768458,
      7.63873279537, 13.1944344379, 11.896042354, 10.5830172417, 10.5020942233,
      92.8918587747, 95.003720371, 86.3715926467, 96.0330217672, 82.6389930677,
      968.702906754, 969.463546828, 1001.79726022, 955.047416547, 1044.27458568;

  expected_out_array << -32.7256441441, -36.4394150514, -9.66467612263,
      -36.4394150514, -36.4394150514, -1.0891900302, -2.66351229645,
      -2.48666868596, -0.929700494428, -3.56327722764, -0.455320135314,
      -0.391437214323, -0.491352055991, -0.350454834292, -0.471773162921,
      -0.104084440522, -0.0723646747909, -0.0992828975532, -0.121638215446,
      -0.122619605294, -0.0317670267286, -0.0359974812869, -0.0154359225363,
      -0.0375775365921, -0.00794899153653, -0.00777303219211, -0.00796085782042,
      -0.0125850719397, -0.00455500206958, -0.00476436993148;

  for (int i = 0; i < 30; ++i) {
    in_x(i) = in_x_array(i);
    in_a(i) = in_a_array(i);
    expected_out(i) = expected_out_array(i);
  }

  std::size_t bytes = in_x.size() * sizeof(Scalar);

  Scalar* d_a;
  Scalar* d_x;
  Scalar* d_out;
  gpuMalloc((void**)(&d_a), bytes);
  gpuMalloc((void**)(&d_x), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_a, in_a.data(), bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_x, in_x.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_a(d_a, 30);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_x(d_x, 30);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 30);

  gpu_out.device(gpu_device) = gpu_a.igamma_der_a(gpu_x);

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost,
                         gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 30; ++i) {
    VERIFY_IS_APPROX(out(i), expected_out(i));
  }

  gpuFree(d_a);
  gpuFree(d_x);
  gpuFree(d_out);
}

template <typename Scalar>
void test_gpu_gamma_sample_der_alpha()
{
  Tensor<Scalar, 1> in_alpha(30);
  Tensor<Scalar, 1> in_sample(30);
  Tensor<Scalar, 1> out(30);
  Tensor<Scalar, 1> expected_out(30);
  out.setZero();

  Array<Scalar, 1, Dynamic> in_alpha_array(30);
  Array<Scalar, 1, Dynamic> in_sample_array(30);
  Array<Scalar, 1, Dynamic> expected_out_array(30);

  // See special_functions.cpp for the Python code that generates the test data.

  in_alpha_array << 0.01, 0.01, 0.01, 0.01, 0.01, 0.1, 0.1, 0.1, 0.1, 0.1, 1.0,
      1.0, 1.0, 1.0, 1.0, 10.0, 10.0, 10.0, 10.0, 10.0, 100.0, 100.0, 100.0,
      100.0, 100.0, 1000.0, 1000.0, 1000.0, 1000.0, 1000.0;

  in_sample_array << 1.25668890405e-26, 1.17549435082e-38, 1.20938905072e-05,
      1.17549435082e-38, 1.17549435082e-38, 5.66572070696e-16, 0.0132865061065,
      0.0200034203853, 6.29263709118e-17, 1.37160367764e-06, 0.333412038288,
      1.18135687766, 0.580629033777, 0.170631439426, 0.786686768458,
      7.63873279537, 13.1944344379, 11.896042354, 10.5830172417, 10.5020942233,
      92.8918587747, 95.003720371, 86.3715926467, 96.0330217672, 82.6389930677,
      968.702906754, 969.463546828, 1001.79726022, 955.047416547, 1044.27458568;

  expected_out_array << 7.42424742367e-23, 1.02004297287e-34, 0.0130155240738,
      1.02004297287e-34, 1.02004297287e-34, 1.96505168277e-13, 0.525575786243,
      0.713903991771, 2.32077561808e-14, 0.000179348049886, 0.635500453302,
      1.27561284917, 0.878125852156, 0.41565819538, 1.03606488534,
      0.885964824887, 1.16424049334, 1.10764479598, 1.04590810812,
      1.04193666963, 0.965193152414, 0.976217589464, 0.93008035061,
      0.98153216096, 0.909196397698, 0.98434963993, 0.984738050206,
      1.00106492525, 0.97734200649, 1.02198794179;

  for (int i = 0; i < 30; ++i) {
    in_alpha(i) = in_alpha_array(i);
    in_sample(i) = in_sample_array(i);
    expected_out(i) = expected_out_array(i);
  }

  std::size_t bytes = in_alpha.size() * sizeof(Scalar);

  Scalar* d_alpha;
  Scalar* d_sample;
  Scalar* d_out;
  gpuMalloc((void**)(&d_alpha), bytes);
  gpuMalloc((void**)(&d_sample), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_alpha, in_alpha.data(), bytes, gpuMemcpyHostToDevice);
  gpuMemcpy(d_sample, in_sample.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_alpha(d_alpha, 30);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_sample(d_sample, 30);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 30);

  gpu_out.device(gpu_device) = gpu_alpha.gamma_sample_der_alpha(gpu_sample);

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost,
                         gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 30; ++i) {
    VERIFY_IS_APPROX(out(i), expected_out(i));
  }

  gpuFree(d_alpha);
  gpuFree(d_sample);
  gpuFree(d_out);
}

template <typename Scalar>
void test_gpu_dawsn()
{
  Tensor<Scalar, 1> in_x(60);
  Tensor<Scalar, 1> out(60);
  Tensor<Scalar, 1> expected_out(60);
  out.setZero();

  Array<Scalar, 1, Dynamic> in_x_array(60);
  Array<Scalar, 1, Dynamic> expected_out_array(60);

  // Compare against scipy.special.dawsn.

  in_x_array << -30.        , -28.98305085, -27.96610169, -26.94915254,
       -25.93220339, -24.91525424, -23.89830508, -22.88135593,
       -21.86440678, -20.84745763, -19.83050847, -18.81355932,
       -17.79661017, -16.77966102, -15.76271186, -14.74576271,
       -13.72881356, -12.71186441, -11.69491525, -10.6779661 ,
        -9.66101695,  -8.6440678 ,  -7.62711864,  -6.61016949,
        -5.59322034,  -4.57627119,  -3.55932203,  -2.54237288,
        -1.52542373,  -0.50847458,   0.50847458,   1.52542373,
         2.54237288,   3.55932203,   4.57627119,   5.59322034,
         6.61016949,   7.62711864,   8.6440678 ,   9.66101695,
        10.6779661 ,  11.69491525,  12.71186441,  13.72881356,
        14.74576271,  15.76271186,  16.77966102,  17.79661017,
        18.81355932,  19.83050847,  20.84745763,  21.86440678,
        22.88135593,  23.89830508,  24.91525424,  25.93220339,
        26.94915254,  27.96610169,  28.98305085,  30.;
  expected_out_array << -0.01667594, -0.01726175, -0.01789024, -0.01856626, -0.01929541,
      -0.02008423, -0.02094035, -0.02187278, -0.02289221, -0.02401143,
      -0.02524586, -0.02661428, -0.0281398 , -0.02985118, -0.03178465,
      -0.03398656, -0.03651715, -0.03945619, -0.04291167, -0.0470335 ,
      -0.05203622, -0.05823825, -0.0661342 , -0.07653816, -0.09089738,
      -0.11208184, -0.14686808, -0.21830822, -0.42101208, -0.4292628 ,
       0.4292628 ,  0.42101208,  0.21830822,  0.14686808,  0.11208184,
       0.09089738,  0.07653816,  0.0661342 ,  0.05823825,  0.05203622,
       0.0470335 ,  0.04291167,  0.03945619,  0.03651715,  0.03398656,
       0.03178465,  0.02985118,  0.0281398 ,  0.02661428,  0.02524586,
       0.02401143,  0.02289221,  0.02187278,  0.02094035,  0.02008423,
       0.01929541,  0.01856626,  0.01789024,  0.01726175,  0.01667594;


  for (int i = 0; i < 60; ++i) {
    in_x(i) = in_x_array(i);
    expected_out(i) = expected_out_array(i);
  }

  std::size_t bytes = in_x.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out;
  gpuMalloc((void**)(&d_in), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_in, in_x.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in(d_in, 60);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 60);

  gpu_out.device(gpu_device) = gpu_in.dawsn();

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost,
                         gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 60; ++i) {
    VERIFY_IS_APPROX(out(i), expected_out(i));
  }

  gpuFree(d_in);
  gpuFree(d_out);
}

template <typename Scalar>
void test_gpu_expi()
{
  Tensor<Scalar, 1> in_x(83);
  Tensor<Scalar, 1> out(83);
  Tensor<Scalar, 1> expected_out(83);
  out.setZero();

  Array<Scalar, 1, Dynamic> in_x_array(83);
  Array<Scalar, 1, Dynamic> expected_out_array(83);

  in_x_array <<  0.        ,   1.07563025,   2.1512605 ,   3.22689076,
         4.30252101,   5.37815126,   6.45378151,   7.52941176,
         8.60504202,   9.68067227,  10.75630252,  11.83193277,
        12.90756303,  13.98319328,  15.05882353,  16.13445378,
        17.21008403,  18.28571429,  19.36134454,  20.43697479,
        21.51260504,  22.58823529,  23.66386555,  24.7394958 ,
        25.81512605,  26.8907563 ,  27.96638655,  29.04201681,
        30.11764706,  31.19327731,  32.26890756,  33.34453782,
        34.42016807,  35.49579832,  36.57142857,  37.64705882,
        38.72268908,  39.79831933,  40.87394958,  41.94957983,
        43.02521008,  44.10084034,  45.17647059,  46.25210084,
        47.32773109,  48.40336134,  49.4789916 ,  50.55462185,
        51.6302521 ,  52.70588235,  53.78151261,  54.85714286,
        55.93277311,  57.00840336,  58.08403361,  59.15966387,
        60.23529412,  61.31092437,  62.38655462,  63.46218487,
        64.53781513,  65.61344538,  66.68907563,  67.76470588,
        68.84033613,  69.91596639,  70.99159664,  72.06722689,
        73.14285714,  74.21848739,  75.29411765,  76.3697479 ,
        77.44537815,  78.5210084 ,  79.59663866,  80.67226891,
        81.74789916,  82.82352941,  83.89915966,  84.97478992,
        86.05042017,  87.12605042,  88.20168067;

  expected_out_array <<   -plusinf, 2.10089121e+00, 5.53528985e+00, 1.15753783e+01,
       2.42706304e+01, 5.33055760e+01, 1.23108954e+02, 2.96569919e+02,
       7.37905242e+02, 1.88081172e+03, 4.88150242e+03, 1.28468967e+04,
       3.41823396e+04, 9.17600177e+04, 2.48134005e+05, 6.75145588e+05,
       1.84670692e+06, 5.07434607e+06, 1.39989488e+07, 3.87561319e+07,
       1.07633111e+08, 2.99757166e+08, 8.36932884e+08, 2.34210043e+09,
       6.56787329e+09, 1.84531523e+10, 5.19367514e+10, 1.46412247e+11,
       4.13357022e+11, 1.16861448e+12, 3.30805712e+12, 9.37546109e+12,
       2.66008817e+13, 7.55531072e+13, 2.14799634e+14, 6.11243402e+14,
       1.74088881e+15, 4.96229772e+15, 1.41556570e+16, 4.04105470e+16,
       1.15441080e+17, 3.29998307e+17, 9.43918678e+17, 2.70156782e+18,
       7.73648003e+18, 2.21669450e+19, 6.35466415e+19, 1.82261026e+20,
       5.22998247e+20, 1.50142352e+21, 4.31215780e+21, 1.23898742e+22,
       3.56133479e+22, 1.02406049e+23, 2.94577085e+23, 8.47670055e+23,
       2.44007728e+24, 7.02625342e+24, 2.02386922e+25, 5.83142468e+25,
       1.68072212e+26, 4.84553619e+26, 1.39736005e+27, 4.03080178e+27,
       1.16302074e+28, 3.35654768e+28, 9.68956171e+28, 2.79780963e+29,
       8.08038054e+29, 2.33422190e+30, 6.74444651e+30, 1.94913373e+31,
       5.63411260e+31, 1.62890401e+32, 4.71030651e+32, 1.36233614e+33,
       3.94092917e+33, 1.14022387e+34, 3.29956549e+34, 9.54985039e+34,
       2.76444343e+35, 8.00365528e+35, 2.31759127e+36;



  for (int i = 0; i < 83; ++i) {
    in_x(i) = in_x_array(i);
    expected_out(i) = expected_out_array(i);
  }

  std::size_t bytes = in_x.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out;
  gpuMalloc((void**)(&d_in), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_in, in_x.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in(d_in, 83);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 83);

  gpu_out.device(gpu_device) = gpu_in.expi();

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost,
                         gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 83; ++i) {
    VERIFY_IS_APPROX(out(i), expected_out(i));
  }

  gpuFree(d_in);
  gpuFree(d_out);
}

template <typename Scalar>
void test_gpu_fresnel()
{
  Tensor<Scalar, 1> in_x(120);
  Tensor<Scalar, 1> out1(120);
  Tensor<Scalar, 1> expected_out1(120);
  Tensor<Scalar, 1> out2(120);
  Tensor<Scalar, 1> expected_out2(120);
  out.setZero();

  Array<Scalar, 1, Dynamic> in_x_array(120);
  Array<Scalar, 1, Dynamic> expected_out1_array(120);
  Array<Scalar, 1, Dynamic> expected_out2_array(120);

  in_x_array << 0.        ,   0.84033613,   1.68067227,   2.5210084 ,
         3.36134454,   4.20168067,   5.04201681,   5.88235294,
         6.72268908,   7.56302521,   8.40336134,   9.24369748,
        10.08403361,  10.92436975,  11.76470588,  12.60504202,
        13.44537815,  14.28571429,  15.12605042,  15.96638655,
        16.80672269,  17.64705882,  18.48739496,  19.32773109,
        20.16806723,  21.00840336,  21.8487395 ,  22.68907563,
        23.52941176,  24.3697479 ,  25.21008403,  26.05042017,
        26.8907563 ,  27.73109244,  28.57142857,  29.41176471,
        30.25210084,  31.09243697,  31.93277311,  32.77310924,
        33.61344538,  34.45378151,  35.29411765,  36.13445378,
        36.97478992,  37.81512605,  38.65546218,  39.49579832,
        40.33613445,  41.17647059,  42.01680672,  42.85714286,
        43.69747899,  44.53781513,  45.37815126,  46.21848739,
        47.05882353,  47.89915966,  48.7394958 ,  49.57983193,
        50.42016807,  51.2605042 ,  52.10084034,  52.94117647,
        53.78151261,  54.62184874,  55.46218487,  56.30252101,
        57.14285714,  57.98319328,  58.82352941,  59.66386555,
        60.50420168,  61.34453782,  62.18487395,  63.02521008,
        63.86554622,  64.70588235,  65.54621849,  66.38655462,
        67.22689076,  68.06722689,  68.90756303,  69.74789916,
        70.58823529,  71.42857143,  72.26890756,  73.1092437 ,
        73.94957983,  74.78991597,  75.6302521 ,  76.47058824,
        77.31092437,  78.1512605 ,  78.99159664,  79.83193277,
        80.67226891,  81.51260504,  82.35294118,  83.19327731,
        84.03361345,  84.87394958,  85.71428571,  86.55462185,
        87.39495798,  88.23529412,  89.07563025,  89.91596639,
        90.75630252,  91.59663866,  92.43697479,  93.27731092,
        94.11764706,  94.95798319,  95.79831933,  96.63865546,
        97.4789916 ,  98.31932773,  99.15966387, 100.;

  expected_out1_array << 0.        , 0.74266522, 0.32812213, 0.43875588, 0.41453588,
       0.54029029, 0.55022786, 0.45642045, 0.54524838, 0.54010938,
       0.46888788, 0.5264084 , 0.51495646, 0.47492414, 0.48386262,
       0.47515281, 0.52223741, 0.50281483, 0.51997787, 0.4802031 ,
       0.48735932, 0.4857135 , 0.50575129, 0.51048487, 0.48541441,
       0.51288696, 0.51221434, 0.48670058, 0.50737654, 0.50236137,
       0.49177003, 0.48984997, 0.48834718, 0.51147597, 0.50546322,
       0.51078691, 0.48994025, 0.49060812, 0.49549969, 0.49883568,
       0.50201447, 0.49080629, 0.50441198, 0.50401653, 0.49158384,
       0.5002166 , 0.49691317, 0.49896422, 0.4921087 , 0.49454764,
       0.5060444 , 0.50679103, 0.50539112, 0.49595384, 0.4932533 ,
       0.50159178, 0.49497724, 0.49671284, 0.49566851, 0.49840677,
       0.49811273, 0.49666623, 0.49569701, 0.49438201, 0.50385107,
       0.49618958, 0.5004852 , 0.50023254, 0.50493898, 0.49956376,
       0.50173312, 0.49816757, 0.50488652, 0.49495883, 0.49489211,
       0.5013866 , 0.49524103, 0.4952145 , 0.50225019, 0.49538438,
       0.49642285, 0.50455173, 0.50178249, 0.50426757, 0.49598558,
       0.49971476, 0.49582196, 0.50434592, 0.50323022, 0.50285698,
       0.49957101, 0.49841209, 0.50411503, 0.49770791, 0.49801616,
       0.50389467, 0.50009259, 0.50179876, 0.49995817, 0.50375684,
       0.50198849, 0.49773544, 0.49630356, 0.49834349, 0.50069004,
       0.50267925, 0.49760382, 0.5034784 , 0.50314093, 0.50030427,
       0.50276759, 0.50292828, 0.49930655, 0.50335069, 0.50291682,
       0.49670977, 0.49921886, 0.49713836, 0.50270743, 0.4999999;

  expected_out2_array << 0.        , 0.28445073, 0.56803166, 0.60957677, 0.45964101,
       0.56408363, 0.5382052 , 0.53205885, 0.51392546, 0.51274102,
       0.52160323, 0.52209767, 0.5277966 , 0.48516185, 0.52171661,
       0.50450528, 0.49187834, 0.47789696, 0.49338769, 0.50235243,
       0.51410366, 0.48898874, 0.51622867, 0.51270028, 0.50602981,
       0.50796839, 0.50794099, 0.50446585, 0.51134009, 0.51284645,
       0.49042447, 0.50680298, 0.49791918, 0.50023842, 0.49029065,
       0.50087741, 0.49691587, 0.50407426, 0.49110558, 0.50964249,
       0.50925297, 0.49908881, 0.50786592, 0.50784007, 0.49818886,
       0.50841474, 0.50763407, 0.4920075 , 0.49995515, 0.49451998,
       0.50456702, 0.4969924 , 0.5048988 , 0.49410869, 0.49807991,
       0.49329941, 0.50453042, 0.50577548, 0.49511225, 0.50621932,
       0.50602445, 0.49476113, 0.50433707, 0.50214211, 0.49550569,
       0.49559084, 0.49428132, 0.50564878, 0.50257605, 0.50547233,
       0.49487378, 0.49498951, 0.49805076, 0.49877073, 0.50033348,
       0.49514355, 0.50148089, 0.50113966, 0.49569652, 0.49870148,
       0.49689792, 0.50107262, 0.49573839, 0.49838279, 0.50205403,
       0.5044472 , 0.50139418, 0.49973669, 0.49715506, 0.50315462,
       0.49581316, 0.49615226, 0.49986416, 0.49663316, 0.49649249,
       0.50085425, 0.49605537, 0.49653391, 0.50386497, 0.50072498,
       0.50322397, 0.49701049, 0.50035675, 0.49671664, 0.50357624,
       0.50241573, 0.50265105, 0.49934204, 0.49843933, 0.50346178,
       0.49795101, 0.49824773, 0.50331019, 0.50009766, 0.50159141,
       0.49984635, 0.50317061, 0.5015141 , 0.49827535, 0.4968169;


  for (int i = 0; i < 120; ++i) {
    in_x(i) = in_x_array(i);
    expected_out1(i) = expected_out1_array(i);
    expected_out2(i) = expected_out2_array(i);
  }

  std::size_t bytes = in_x.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out1;
  Scalar* d_out2;
  gpuMalloc((void**)(&d_in), bytes);
  gpuMalloc((void**)(&d_out1), bytes);
  gpuMalloc((void**)(&d_out2), bytes);

  gpuMemcpy(d_in, in_x.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in(d_in, 120);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out1(d_out1, 120);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out2(d_out2, 120);

  gpu_out1.device(gpu_device) = gpu_in.fresnel_cos();
  gpu_out2.device(gpu_device) = gpu_in.fresnel_sin();

  assert(gpuMemcpyAsync(out1.data(), d_out1, bytes, gpuMemcpyDeviceToHost,
                         gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);
  assert(gpuMemcpyAsync(out2.data(), d_out2, bytes, gpuMemcpyDeviceToHost,
                         gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 120; ++i) {
    VERIFY_IS_APPROX(out1(i), expected_out1(i));
    VERIFY_IS_APPROX(out2(i), expected_out2(i));
  }

  gpuFree(d_in);
  gpuFree(d_out1);
  gpuFree(d_out2);
}

template <typename Scalar>
void test_gpu_spence()
{
  Tensor<Scalar, 1> in_x(120);
  Tensor<Scalar, 1> out(120);
  Tensor<Scalar, 1> expected_out(120);
  out.setZero();

  Array<Scalar, 1, Dynamic> in_x_array(120);
  Array<Scalar, 1, Dynamic> expected_out_array(120);

  in_x_array <<   0.        ,   1.07563025,   2.1512605 ,   3.22689076,
         4.30252101,   5.37815126,   6.45378151,   7.52941176,
         8.60504202,   9.68067227,  10.75630252,  11.83193277,
        12.90756303,  13.98319328,  15.05882353,  16.13445378,
        17.21008403,  18.28571429,  19.36134454,  20.43697479,
        21.51260504,  22.58823529,  23.66386555,  24.7394958 ,
        25.81512605,  26.8907563 ,  27.96638655,  29.04201681,
        30.11764706,  31.19327731,  32.26890756,  33.34453782,
        34.42016807,  35.49579832,  36.57142857,  37.64705882,
        38.72268908,  39.79831933,  40.87394958,  41.94957983,
        43.02521008,  44.10084034,  45.17647059,  46.25210084,
        47.32773109,  48.40336134,  49.4789916 ,  50.55462185,
        51.6302521 ,  52.70588235,  53.78151261,  54.85714286,
        55.93277311,  57.00840336,  58.08403361,  59.15966387,
        60.23529412,  61.31092437,  62.38655462,  63.46218487,
        64.53781513,  65.61344538,  66.68907563,  67.76470588,
        68.84033613,  69.91596639,  70.99159664,  72.06722689,
        73.14285714,  74.21848739,  75.29411765,  76.3697479 ,
        77.44537815,  78.5210084 ,  79.59663866,  80.67226891,
        81.74789916,  82.82352941,  83.89915966,  84.97478992,
        86.05042017,  87.12605042,  88.20168067,  89.27731092,
        90.35294118,  91.42857143,  92.50420168,  93.57983193,
        94.65546218,  95.73109244,  96.80672269,  97.88235294,
        98.95798319, 100.03361345, 101.1092437 , 102.18487395,
       103.2605042 , 104.33613445, 105.41176471, 106.48739496,
       107.56302521, 108.63865546, 109.71428571, 110.78991597,
       111.86554622, 112.94117647, 114.01680672, 115.09243697,
       116.16806723, 117.24369748, 118.31932773, 119.39495798,
       120.47058824, 121.54621849, 122.62184874, 123.69747899,
       124.77310924, 125.8487395 , 126.92436975, 128.;

  expected_out_array <<   1.64493407,  -0.07424638,  -0.92517856,  -1.55869701,
        -2.07605073,  -2.5186072 ,  -2.908091  ,  -3.25755955,
        -3.57556745,  -3.86806807,  -4.13939443,  -4.3928116 ,
        -4.63084902,  -4.85551113,  -5.06841659,  -5.27089367,
        -5.46404732,  -5.64880782,  -5.82596653,  -5.99620293,
        -6.16010524,  -6.31818641,  -6.47089677,  -6.61863403,
        -6.76175137,  -6.900564  ,  -7.03535452,  -7.16637741,
        -7.29386268,  -7.41801901,  -7.53903635,  -7.65708816,
        -7.7723333 ,  -7.88491766,  -7.99497556,  -8.10263099,
        -8.20799864,  -8.31118484,  -8.41228837,  -8.51140117,
        -8.608609  ,  -8.70399192,  -8.79762489,  -8.88957813,
        -8.97991756,  -9.06870512,  -9.15599912,  -9.24185452,
        -9.32632318,  -9.40945407,  -9.49129353,  -9.57188543,
        -9.65127137,  -9.7294908 ,  -9.8065812 ,  -9.8825782 ,
        -9.95751571, -10.03142601, -10.10433989, -10.17628671,
       -10.2472945 , -10.31739005, -10.38659896, -10.45494576,
       -10.5224539 , -10.58914586, -10.65504321, -10.72016663,
       -10.78453597, -10.84817031, -10.91108798, -10.97330659,
       -11.03484312, -11.0957139 , -11.15593464, -11.2155205 ,
       -11.2744861 , -11.33284553, -11.39061239, -11.44779982,
       -11.50442049, -11.56048667, -11.6160102 , -11.67100254,
       -11.72547479, -11.77943766, -11.83290156, -11.88587655,
       -11.93837237, -11.99039848, -12.04196404, -12.09307795,
       -12.14374883, -12.19398505, -12.24379474, -12.2931858 ,
       -12.34216589, -12.39074247, -12.43892277, -12.48671386,
       -12.53412257, -12.58115558, -12.62781937, -12.67412026,
       -12.72006439, -12.76565776, -12.81090619, -12.85581537,
       -12.90039084, -12.94463799, -12.98856211, -13.03216831,
       -13.07546161, -13.1184469 , -13.16112895, -13.20351241,
       -13.24560183, -13.28740166, -13.32891622, -13.37014976;

  for (int i = 0; i < 120; ++i) {
    in_x(i) = in_x_array(i);
    expected_out(i) = expected_out_array(i);
  }

  std::size_t bytes = in_x.size() * sizeof(Scalar);

  Scalar* d_in;
  Scalar* d_out;
  gpuMalloc((void**)(&d_in), bytes);
  gpuMalloc((void**)(&d_out), bytes);

  gpuMemcpy(d_in, in_x.data(), bytes, gpuMemcpyHostToDevice);

  Eigen::GpuStreamDevice stream;
  Eigen::GpuDevice gpu_device(&stream);

  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_in(d_in, 120);
  Eigen::TensorMap<Eigen::Tensor<Scalar, 1> > gpu_out(d_out, 120);

  gpu_out.device(gpu_device) = gpu_in.spence();

  assert(gpuMemcpyAsync(out.data(), d_out, bytes, gpuMemcpyDeviceToHost,
                         gpu_device.stream()) == gpuSuccess);
  assert(gpuStreamSynchronize(gpu_device.stream()) == gpuSuccess);

  for (int i = 0; i < 120; ++i) {
    VERIFY_IS_APPROX(out(i), expected_out(i));
  }

  gpuFree(d_in);
  gpuFree(d_out);
}


EIGEN_DECLARE_TEST(cxx11_tensor_gpu)
{
  CALL_SUBTEST_1(test_gpu_nullary());
  CALL_SUBTEST_1(test_gpu_elementwise_small());
  CALL_SUBTEST_1(test_gpu_elementwise());
  CALL_SUBTEST_1(test_gpu_props());
  CALL_SUBTEST_1(test_gpu_reduction());
  CALL_SUBTEST_2(test_gpu_contraction<ColMajor>());
  CALL_SUBTEST_2(test_gpu_contraction<RowMajor>());
  CALL_SUBTEST_3(test_gpu_convolution_1d<ColMajor>());
  CALL_SUBTEST_3(test_gpu_convolution_1d<RowMajor>());
  CALL_SUBTEST_3(test_gpu_convolution_inner_dim_col_major_1d());
  CALL_SUBTEST_3(test_gpu_convolution_inner_dim_row_major_1d());
  CALL_SUBTEST_3(test_gpu_convolution_2d<ColMajor>());
  CALL_SUBTEST_3(test_gpu_convolution_2d<RowMajor>());
#if !defined(EIGEN_USE_HIP)
// disable these tests on HIP for now.
// they hang..need to investigate and fix
  CALL_SUBTEST_3(test_gpu_convolution_3d<ColMajor>());
  CALL_SUBTEST_3(test_gpu_convolution_3d<RowMajor>());
#endif

#if EIGEN_GPU_TEST_C99_MATH
  // std::erf, std::erfc, and so on where only added in c++11. We use them
  // as a golden reference to validate the results produced by Eigen. Therefore
  // we can only run these tests if we use a c++11 compiler.
  CALL_SUBTEST_4(test_gpu_lgamma<float>(1.0f));
  CALL_SUBTEST_4(test_gpu_lgamma<float>(100.0f));
  CALL_SUBTEST_4(test_gpu_lgamma<float>(0.01f));
  CALL_SUBTEST_4(test_gpu_lgamma<float>(0.001f));

  CALL_SUBTEST_4(test_gpu_lgamma<double>(1.0));
  CALL_SUBTEST_4(test_gpu_lgamma<double>(100.0));
  CALL_SUBTEST_4(test_gpu_lgamma<double>(0.01));
  CALL_SUBTEST_4(test_gpu_lgamma<double>(0.001));

  CALL_SUBTEST_4(test_gpu_erf<float>(1.0f));
  CALL_SUBTEST_4(test_gpu_erf<float>(100.0f));
  CALL_SUBTEST_4(test_gpu_erf<float>(0.01f));
  CALL_SUBTEST_4(test_gpu_erf<float>(0.001f));

  CALL_SUBTEST_4(test_gpu_erfc<float>(1.0f));
  // CALL_SUBTEST(test_gpu_erfc<float>(100.0f));
  CALL_SUBTEST_4(test_gpu_erfc<float>(5.0f)); // GPU erfc lacks precision for large inputs
  CALL_SUBTEST_4(test_gpu_erfc<float>(0.01f));
  CALL_SUBTEST_4(test_gpu_erfc<float>(0.001f));

  CALL_SUBTEST_4(test_gpu_erf<double>(1.0));
  CALL_SUBTEST_4(test_gpu_erf<double>(100.0));
  CALL_SUBTEST_4(test_gpu_erf<double>(0.01));
  CALL_SUBTEST_4(test_gpu_erf<double>(0.001));

  CALL_SUBTEST_4(test_gpu_erfc<double>(1.0));
  // CALL_SUBTEST(test_gpu_erfc<double>(100.0));
  CALL_SUBTEST_4(test_gpu_erfc<double>(5.0)); // GPU erfc lacks precision for large inputs
  CALL_SUBTEST_4(test_gpu_erfc<double>(0.01));
  CALL_SUBTEST_4(test_gpu_erfc<double>(0.001));

#if !defined(EIGEN_USE_HIP)
// disable these tests on HIP for now.

  CALL_SUBTEST_5(test_gpu_ndtri<float>());
  CALL_SUBTEST_5(test_gpu_ndtri<double>());

  CALL_SUBTEST_5(test_gpu_digamma<float>());
  CALL_SUBTEST_5(test_gpu_digamma<double>());

  CALL_SUBTEST_5(test_gpu_polygamma<float>());
  CALL_SUBTEST_5(test_gpu_polygamma<double>());

  CALL_SUBTEST_5(test_gpu_zeta<float>());
  CALL_SUBTEST_5(test_gpu_zeta<double>());
#endif

  CALL_SUBTEST_5(test_gpu_igamma<float>());
  CALL_SUBTEST_5(test_gpu_igammac<float>());

  CALL_SUBTEST_5(test_gpu_igamma<double>());
  CALL_SUBTEST_5(test_gpu_igammac<double>());

#if !defined(EIGEN_USE_HIP)
// disable these tests on HIP for now.
  CALL_SUBTEST_6(test_gpu_betainc<float>());
  CALL_SUBTEST_6(test_gpu_betainc<double>());

  CALL_SUBTEST_6(test_gpu_i0e<float>());
  CALL_SUBTEST_6(test_gpu_i0e<double>());

  CALL_SUBTEST_6(test_gpu_i1e<float>());
  CALL_SUBTEST_6(test_gpu_i1e<double>());

  CALL_SUBTEST_6(test_gpu_i1e<float>());
  CALL_SUBTEST_6(test_gpu_i1e<double>());

  CALL_SUBTEST_6(test_gpu_igamma_der_a<float>());
  CALL_SUBTEST_6(test_gpu_igamma_der_a<double>());

  CALL_SUBTEST_6(test_gpu_gamma_sample_der_alpha<float>());
  CALL_SUBTEST_6(test_gpu_gamma_sample_der_alpha<double>());
#endif

#if !defined(EIGEN_USE_HIP)
// disable these tests on HIP for now.
  CALL_SUBTEST_7(test_gpu_dawsn<float>());
  CALL_SUBTEST_7(test_gpu_dawsn<double>());

  CALL_SUBTEST_7(test_gpu_expi<float>());
  CALL_SUBTEST_7(test_gpu_expi<double>());

  CALL_SUBTEST_7(test_gpu_fresnel<float>());
  CALL_SUBTEST_7(test_gpu_fresnel<double>());

  CALL_SUBTEST_7(test_gpu_spence<float>());
  CALL_SUBTEST_7(test_gpu_spence<double>());
#endif


#endif
}
